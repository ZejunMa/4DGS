/*
 * Copyright (C) 2023, Inria
 * GRAPHDECO research group, https://team.inria.fr/graphdeco
 * All rights reserved.
 *
 * This software is free for non-commercial, research and evaluation use 
 * under the terms of the LICENSE.md file.
 *
 * For inquiries contact  george.drettakis@inria.fr
 */

#include <math.h>
#include <torch/extension.h>
#include <cstdio>
#include <sstream>
#include <iostream>
#include <tuple>
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <memory>
#include "cuda_rasterizer/config.h"
#include "cuda_rasterizer/rasterizer.h"
#include "cuda_rasterizer/rasterizer_impl.h"
#include <fstream>
#include <string>
#include <functional>

std::function<char*(size_t N)> resizeFunctional(torch::Tensor& t) {
    auto lambda = [&t](size_t N) {
        t.resize_({(long long)N});
		return reinterpret_cast<char*>(t.contiguous().data_ptr());
    };
    return lambda;
}

std::tuple<int, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor>
RasterizeGaussiansCUDA(
	const torch::Tensor& background,
	const torch::Tensor& means3D,
    const torch::Tensor& colors,
	const torch::Tensor& flows,
    const torch::Tensor& opacity,
	const torch::Tensor& ts,
	const torch::Tensor& scales,
	const torch::Tensor& scales_t,
	const torch::Tensor& rotations,
	const torch::Tensor& rotations_r,
	const float scale_modifier,
	const torch::Tensor& cov3D_precomp,
	const torch::Tensor& viewmatrix,
	const torch::Tensor& projmatrix,
	const float tan_fovx,
	const float tan_fovy,
    const int image_height,
    const int image_width,
	const torch::Tensor& sh,
	const int degree,
	const int degree_t, 
	const torch::Tensor& campos,
	const float timestamp,
	const float time_duration,
	const bool rot_4d,
	const int gaussian_dim,
	const bool force_sh_3d,
	const bool prefiltered,
	const bool debug)
{
  if (means3D.ndimension() != 2 || means3D.size(1) != 3) {
    AT_ERROR("means3D must have dimensions (num_points, 3)");
  }
  
  const int P = means3D.size(0);
  const int H = image_height;
  const int W = image_width;

  auto int_opts = means3D.options().dtype(torch::kInt32);
  auto float_opts = means3D.options().dtype(torch::kFloat32);

  torch::Tensor out_color = torch::full({NUM_CHANNELS, H, W}, 0.0, float_opts);
  torch::Tensor out_flow = torch::full({2, H, W}, 0.0, float_opts);
  torch::Tensor out_depth = torch::full({1, H, W}, 0.0, float_opts);
  torch::Tensor out_T = torch::full({1, H, W}, 0.0, float_opts);
  torch::Tensor radii = torch::full({P}, 0, means3D.options().dtype(torch::kInt32));
  
  torch::Device device(torch::kCUDA);
  torch::TensorOptions options(torch::kByte);
  torch::Tensor geomBuffer = torch::empty({0}, options.device(device));
  torch::Tensor binningBuffer = torch::empty({0}, options.device(device));
  torch::Tensor imgBuffer = torch::empty({0}, options.device(device));
  std::function<char*(size_t)> geomFunc = resizeFunctional(geomBuffer);
  std::function<char*(size_t)> binningFunc = resizeFunctional(binningBuffer);
  std::function<char*(size_t)> imgFunc = resizeFunctional(imgBuffer);
  
  int rendered = 0;
  if(P != 0)
  {
	  int M = 0;
	  if(sh.size(0) != 0)
	  {
		M = sh.size(1);
      }

	  rendered = CudaRasterizer::Rasterizer::forward(
	    geomFunc,
		binningFunc,
		imgFunc,
	    P, degree, degree_t, M,
		background.contiguous().data<float>(),
		W, H,
		means3D.contiguous().data<float>(),
		sh.contiguous().data_ptr<float>(),
		colors.contiguous().data<float>(), 
		flows.contiguous().data<float>(),
		opacity.contiguous().data<float>(), 
		ts.contiguous().data_ptr<float>(), 
		scales.contiguous().data_ptr<float>(),
		scales_t.contiguous().data_ptr<float>(), 
		scale_modifier,
		rotations.contiguous().data_ptr<float>(),
		rotations_r.contiguous().data_ptr<float>(),
		cov3D_precomp.contiguous().data<float>(), 
		viewmatrix.contiguous().data<float>(), 
		projmatrix.contiguous().data<float>(),
		campos.contiguous().data<float>(),
		timestamp,
		time_duration,
		rot_4d,
		gaussian_dim,
		force_sh_3d,
		tan_fovx,
		tan_fovy,
		prefiltered,
		out_color.contiguous().data<float>(),
		out_flow.contiguous().data<float>(), 
		out_depth.contiguous().data<float>(),
		out_T.contiguous().data<float>(),
		radii.contiguous().data<int>(),
		debug);
  }
  char* geo_ptr = reinterpret_cast<char*>(geomBuffer.contiguous().data_ptr());
  CudaRasterizer::GeometryState geoState = CudaRasterizer::GeometryState::fromChunk(geo_ptr, P);

  torch::Tensor covs3D_com = torch::from_blob(geoState.cov3D, {P, 6}, float_opts);
  return std::make_tuple(rendered, out_color, out_flow, out_depth, out_T, radii, geomBuffer, binningBuffer, imgBuffer, covs3D_com);
}

std::tuple<torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor>
 RasterizeGaussiansBackwardCUDA(
 	const torch::Tensor& background,
	const torch::Tensor& means3D,
	const torch::Tensor& radii,
    const torch::Tensor& colors,
	const torch::Tensor& flows_2d,
    const torch::Tensor& opacities,
    const torch::Tensor& ts,
	const torch::Tensor& scales,
	const torch::Tensor& scales_t,
	const torch::Tensor& rotations,
	const torch::Tensor& rotations_r,
	const float scale_modifier,
	const torch::Tensor& cov3D_precomp,
	const torch::Tensor& viewmatrix,
    const torch::Tensor& projmatrix,
	const float tan_fovx,
	const float tan_fovy,
    const torch::Tensor& dL_dout_color,
	const torch::Tensor& dL_dout_depth,
	const torch::Tensor& dL_dout_mask,
	const torch::Tensor& dL_dout_flow,
	const torch::Tensor& sh,
	const int degree,
	const int degree_t,
	const torch::Tensor& campos,
	const float timestamp,
	const float time_duration,
	const bool rot_4d,
	const int gaussian_dim,
	const bool force_sh_3d,
	const torch::Tensor& geomBuffer,
	const int R,
	const torch::Tensor& binningBuffer,
	const torch::Tensor& imageBuffer,
	const bool debug) 
{
  const int P = means3D.size(0);
  const int H = dL_dout_color.size(1);
  const int W = dL_dout_color.size(2);
  
  int M = 0;
  if(sh.size(0) != 0)
  {	
	M = sh.size(1);
  }

  torch::Tensor dL_dmeans3D = torch::zeros({P, 3}, means3D.options());
  torch::Tensor dL_dmeans2D = torch::zeros({P, 3}, means3D.options());
  torch::Tensor dL_dcolors = torch::zeros({P, NUM_CHANNELS}, means3D.options());
  torch::Tensor dL_dflows = torch::zeros({P, 2}, means3D.options());
  torch::Tensor dL_dconic = torch::zeros({P, 2, 2}, means3D.options());
  torch::Tensor dL_dopacity = torch::zeros({P, 1}, means3D.options());
  torch::Tensor dL_dts = torch::zeros({P, 1}, means3D.options());
  torch::Tensor dL_dcov3D = torch::zeros({P, 6}, means3D.options());
  torch::Tensor dL_dsh = torch::zeros({P, M, 3}, means3D.options());
  torch::Tensor dL_dscales = torch::zeros({P, 3}, means3D.options());
  torch::Tensor dL_dscales_t = torch::zeros({P, 1}, means3D.options());
  torch::Tensor dL_drotations = torch::zeros({P, 4}, means3D.options());
  torch::Tensor dL_drotations_r = torch::zeros({P, 4}, means3D.options());
  
  if(P != 0)
  {  
	  CudaRasterizer::Rasterizer::backward(P, degree, degree_t, M, R,
	  background.contiguous().data<float>(),
	  W, H, 
	  means3D.contiguous().data<float>(),
	  sh.contiguous().data<float>(),
	  colors.contiguous().data<float>(),
	  flows_2d.contiguous().data<float>(),
	  opacities.contiguous().data<float>(),
	  ts.contiguous().data<float>(),
	  scales.data_ptr<float>(),
	  scales_t.data_ptr<float>(),
	  scale_modifier,
	  rotations.data_ptr<float>(),
	  rotations_r.data_ptr<float>(),
	  cov3D_precomp.contiguous().data<float>(),
	  viewmatrix.contiguous().data<float>(),
	  projmatrix.contiguous().data<float>(),
	  campos.contiguous().data<float>(),
	  timestamp,
      time_duration,
      rot_4d,
      gaussian_dim,
      force_sh_3d,
	  tan_fovx,
	  tan_fovy,
	  radii.contiguous().data<int>(),
	  reinterpret_cast<char*>(geomBuffer.contiguous().data_ptr()),
	  reinterpret_cast<char*>(binningBuffer.contiguous().data_ptr()),
	  reinterpret_cast<char*>(imageBuffer.contiguous().data_ptr()),
	  dL_dout_color.contiguous().data<float>(),
	  dL_dout_depth.contiguous().data<float>(),
	  dL_dout_mask.contiguous().data<float>(),
	  dL_dout_flow.contiguous().data<float>(),
	  dL_dmeans2D.contiguous().data<float>(),
	  dL_dconic.contiguous().data<float>(),  
	  dL_dopacity.contiguous().data<float>(),
	  dL_dcolors.contiguous().data<float>(),
	  dL_dmeans3D.contiguous().data<float>(),
	  dL_dcov3D.contiguous().data<float>(),
	  dL_dsh.contiguous().data<float>(),
	  dL_dflows.contiguous().data<float>(),
	  dL_dts.contiguous().data<float>(),
	  dL_dscales.contiguous().data<float>(),
	  dL_dscales_t.contiguous().data<float>(),
	  dL_drotations.contiguous().data<float>(),
	  dL_drotations_r.contiguous().data<float>(),
	  debug);
  }

  return std::make_tuple(dL_dmeans2D, dL_dcolors, dL_dopacity, dL_dmeans3D, dL_dcov3D,
        dL_dsh, dL_dflows, dL_dts, dL_dscales, dL_dscales_t, dL_drotations, dL_drotations_r);
}

torch::Tensor markVisible(
		torch::Tensor& means3D,
		torch::Tensor& viewmatrix,
		torch::Tensor& projmatrix)
{ 
  const int P = means3D.size(0);
  
  torch::Tensor present = torch::full({P}, false, means3D.options().dtype(at::kBool));
 
  if(P != 0)
  {
	CudaRasterizer::Rasterizer::markVisible(P,
		means3D.contiguous().data<float>(),
		viewmatrix.contiguous().data<float>(),
		projmatrix.contiguous().data<float>(),
		present.contiguous().data<bool>());
  }
  
  return present;
}